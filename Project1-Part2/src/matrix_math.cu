#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "matrix_math.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
 * Check for CUDA errors; print and exit if there was a problem.
 */
void checkCUDAError(const char *msg, int line = -1) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        if (line >= 0) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

#define blockSize 128
dim3 threadsPerBlock(blockSize);
float *dev_mat_a;
float *dev_mat_b;
float *dev_mat_c;

/**
 * Initialize memory, update some globals
 */
void Matrix_Math::initialize(int N) {
	
    int total = N * N;
    dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);
	
    hipMalloc((void**)&dev_mat_a, total * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc dev_mat_a failed!");

    hipMalloc((void**)&dev_mat_b, total * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc dev_mat_b failed!");

    hipMalloc((void**)&dev_mat_c, total * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc dev_mat_c failed!");

}

void Matrix_Math::cleanUp() {
    hipFree(dev_mat_a);
    hipFree(dev_mat_b);
    hipFree(dev_mat_c);
}

__global__ void mat_add(float *A, float *B, float *C, int N) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N*N) {
		C[index] = A[index] + B[index];
	}
}

__global__ void mat_sub(float *A, float *B, float *C, int N) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N*N) {
		C[index] = A[index] - B[index];
	}
}

__global__ void mat_mul(float *A, float *B, float *C, int N) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N*N) {
		int row = index / N;
		int column = index % N;
		C[index] = 0;

		for(int i = 0; i < N; i++) {
			C[index] += A[i + row*N] * B[column + i*N];
		}
	}
}

void Matrix_Math::kernMatAdd(int N, float *hst_mat_a, float *hst_mat_b, float *hst_mat_c) {
	int total = N * N;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(dev_mat_a, hst_mat_a, total * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_b, hst_mat_b, total * sizeof(float), hipMemcpyHostToDevice);
    dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);

	hipEventRecord(start);
	mat_add<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_mat_a, dev_mat_b, dev_mat_c, N);
	hipEventRecord(stop);

	hipMemcpy(hst_mat_c, dev_mat_c, total * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << milliseconds << "ms for adding" << std::endl;
}

void Matrix_Math::kernMatSub(int N, float *hst_mat_a, float *hst_mat_b, float *hst_mat_c) {
    int total = N * N;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(dev_mat_a, hst_mat_a, total * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_b, hst_mat_b, total * sizeof(float), hipMemcpyHostToDevice);
    dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);

	hipEventRecord(start);
	mat_sub<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_mat_a, dev_mat_b, dev_mat_c, N);
	hipEventRecord(stop);

	hipMemcpy(hst_mat_c, dev_mat_c, total * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << milliseconds << "ms for subtracting" << std::endl;
}

void Matrix_Math::kernMatMul(int N, float *hst_mat_a, float *hst_mat_b, float *hst_mat_c) {
    int total = N * N;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(dev_mat_a, hst_mat_a, total * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_b, hst_mat_b, total * sizeof(float), hipMemcpyHostToDevice);
    dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);

	hipEventRecord(start);
	mat_mul<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_mat_a, dev_mat_b, dev_mat_c, N);
	hipEventRecord(stop);

	hipMemcpy(hst_mat_c, dev_mat_c, total * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << milliseconds << "ms for multiplying" << std::endl;
}