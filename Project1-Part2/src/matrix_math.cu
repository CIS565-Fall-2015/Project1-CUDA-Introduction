#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "matrix_math.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


float *dev_matA;
float *dev_matB;
float *dev_matC;

void MatrixCalc::initMats(float *hst_matA,float *hst_matB,int matWidth)
{
	int size = matWidth*matWidth*sizeof(float);

	hipMalloc((void**)&dev_matA, size);
	checkCUDAErrorWithLine("hipMalloc dev_matA failed!");

	hipMalloc((void**)&dev_matB, size);
	checkCUDAErrorWithLine("hipMalloc dev_matB failed!");

	hipMemcpy(dev_matA,hst_matA,size,hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy hst_matA to dev_matA failed!");

	hipMemcpy(dev_matB, hst_matB, size, hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy hst_matB to dev_matB failed!");

	hipMalloc((void**)&dev_matC, size);
	checkCUDAErrorWithLine("hipMalloc dev_matB failed!");
	//TODO later: try seperate malloc and memcpy

}


__global__ void kernMatAdd(float *matA,float *matB,float *matC,int width)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int idx =  tx*width + ty;
	matC[idx] = matA[idx] + matB[idx];
	
}

void MatrixCalc::mat_add(float*A, float*B, float*C,int width)
{
	initMats(A,B,width);//todo later: 5
	dim3 threadsPerBlock(width, width);
	kernMatAdd<<<1,threadsPerBlock>>>(dev_matA,dev_matB,dev_matC,width);
	hipMemcpy(C,dev_matC,width*width*sizeof(float),hipMemcpyDeviceToHost);
	freeMats();
}


__global__ void kernMatSub(float *matA, float *matB, float *matC, int width)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int idx = tx*width + ty;
	matC[idx] = matA[idx] - matB[idx];

}

void MatrixCalc::mat_sub(float*A, float*B, float*C, int width)
{
	initMats(A, B, width);//todo later: 5
	dim3 threadsPerBlock(width, width);
	kernMatSub <<<1, threadsPerBlock >>>(dev_matA, dev_matB, dev_matC, width);
	hipMemcpy(C, dev_matC, width*width*sizeof(float), hipMemcpyDeviceToHost);
	freeMats();
}

__global__ void kernMatMul(float *matA, float *matB, float *matC, int width)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int idx = tx*width + ty;

	float sum = 0;
	for (int i = 0; i < width; i++)
	{
		sum += matA[tx*width + i] * matB[i*width + ty];
	}
	matC[idx] = sum;

}

void MatrixCalc::mat_mul(float*A, float*B, float*C, int width)
{
	initMats(A, B, width);//todo later: 5
	dim3 threadsPerBlock(width, width);
	kernMatMul <<<1, threadsPerBlock >>>(dev_matA, dev_matB, dev_matC, width);
	hipMemcpy(C, dev_matC, width*width*sizeof(float), hipMemcpyDeviceToHost);
	freeMats();
}

void MatrixCalc::freeMats()
{
	hipFree(dev_matA);
	hipFree(dev_matB);
	hipFree(dev_matC);
}