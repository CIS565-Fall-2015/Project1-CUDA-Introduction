#include "hip/hip_runtime.h"
#include "matrix_math.h"

float *hst_mat_A;
float *hst_mat_B;

float *dev_mat_A;
float *dev_mat_B;
float *dev_mat_C;

void CUDA_matrix_math::initialize() {
	hst_mat_A = (float*) malloc(sizeof(float) * 25);
	hst_mat_B = (float*) malloc(sizeof(float) * 25);
	
	hipMalloc((void**)&dev_mat_A, sizeof(float) * 25);
	hipMalloc((void**)&dev_mat_B, sizeof(float) * 25);
	hipMalloc((void**)&dev_mat_C, sizeof(float) * 25);
}

void CUDA_matrix_math::teardown() {
	free(hst_mat_A);
	free(hst_mat_B);

	hipFree(dev_mat_A);
	hipFree(dev_mat_B);
	hipFree(dev_mat_C);
}

__global__ void mat_add(float *A, float *B, float *C) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j * 5;
	C[index] = A[index] + B[index];
}

__global__ void mat_sub(float *A, float *B, float *C) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j * 5;
	C[index] = A[index] - B[index];
}

__global__ void mat_mul(float *A, float *B, float *C) {
	int corner_x = blockIdx.x * blockDim.x;
	int corner_y = blockIdx.y * blockDim.y;
	int index = (corner_x + threadIdx.x) + (corner_y + threadIdx.y) * 5;
	float dot_product = 0.0f;

	// all values are + blockIdx.x * bloxkDim.x + blockIdx.x + blockIdx.y
	// 0  1  2  3  4
	// 5  6  7  8  9
	// 10 11 12 13 14
	// 15 16 17 18 19
	// 20 21 22 23 24
	int local_index = threadIdx.x + threadIdx.y * 5;
	int corner_index = corner_x + corner_y * 5;
	int col_index = local_index % 5 + corner_index;
	int row_index = (local_index / 5) * 5 + corner_index;
	for (int i = 0; i < 5; i++) {
		dot_product += A[row_index] * B[col_index];
		col_index += 5;
		row_index += 1;
	}

	C[index] = dot_product;
}

void CUDA_matrix_math::cuda_mat_add(float *A, float *B, float *C) {
	hipMemcpy(dev_mat_A, A, 25 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_B, B, 25 * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(5, 5);
	dim3 dimGrid(1, 1);
	mat_add <<<dimGrid, dimBlock >>>(dev_mat_A, dev_mat_B, dev_mat_C);

	hipMemcpy(C, dev_mat_C, 25 * sizeof(float), hipMemcpyDeviceToHost);
}

void CUDA_matrix_math::cuda_mat_sub(float *A, float *B, float *C) {
	hipMemcpy(dev_mat_A, A, 25 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_B, B, 25 * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(5, 5);
	dim3 dimGrid(1, 1);
	mat_sub <<<dimGrid, dimBlock >>>(dev_mat_A, dev_mat_B, dev_mat_C);

	hipMemcpy(C, dev_mat_C, 25 * sizeof(float), hipMemcpyDeviceToHost);
}

void CUDA_matrix_math::cuda_mat_mul(float *A, float *B, float *C) {
	hipMemcpy(dev_mat_A, A, 25 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_B, B, 25 * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(5, 5);
	dim3 dimGrid(1, 1);
	mat_mul <<<dimGrid, dimBlock >>>(dev_mat_A, dev_mat_B, dev_mat_C);

	hipMemcpy(C, dev_mat_C, 25 * sizeof(float), hipMemcpyDeviceToHost);
}