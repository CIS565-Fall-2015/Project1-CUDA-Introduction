#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "config.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
 * Check for CUDA errors; print and exit if there was a problem.
 */
void checkCUDAError(const char *msg, int line = -1) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        if (line >= 0) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// ==============================================
// Initialization
// ==============================================

float *devA;
float *devB;
float *devC;

__host__ int dev_init() {
    if (int i = hipMalloc((void**)&devA, ARRAY_MEM_SIZE)) {
        return i;
    }
    if (int i = hipMalloc((void**)&devB, ARRAY_MEM_SIZE)) {
        hipFree(devA);
        return i;
    }
    if (int i = hipMalloc((void**)&devC, ARRAY_MEM_SIZE)) {
        hipFree(devA);
        hipFree(devB);
        return i;
    }
    return 0;
}

__host__ void dev_free() {
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

__host__ void hst_init(float **a, float **b, float **c) {
    *a = (float*)malloc(ARRAY_MEM_SIZE);
    *b = (float*)malloc(ARRAY_MEM_SIZE);
    *c = (float*)malloc(ARRAY_MEM_SIZE);
}

__host__ void hst_free(float *a, float *b, float *c) {
    free(a);
    free(b);
    free(c);
}

// ==============================================
// Kernel wrappers
// ==============================================

__global__ void kernMatAdd(float *a, float *b, float *c) {
    int i = (threadIdx.y * blockDim.x) + threadIdx.x;
    if (i < ARRAY_SIZE*ARRAY_SIZE) {
        c[i] = a[i] + b[i];
    }
}

__global__ void kernMatSub(float *a, float *b, float *c) {
    int i = (threadIdx.y * blockDim.x) + threadIdx.x;
    if (i < ARRAY_SIZE*ARRAY_SIZE) {
        c[i] = a[i] - b[i];
    }
}

__global__ void kernMatMul(float *a, float *b, float *c) {
    int index = (threadIdx.y * blockDim.x) + threadIdx.x;
    if (threadIdx.x >= ARRAY_SIZE || threadIdx.y >= ARRAY_SIZE) { return; }
    int sum = 0;
    for (int k = 0; k < ARRAY_SIZE; k++) {
        int i = (threadIdx.x * ARRAY_SIZE) + k;
        int j = (k * ARRAY_SIZE) + threadIdx.y;
        sum += a[i] + b[j];
    }
    c[index] = sum;
}

// ==============================================
// Kernel wrappers
// ==============================================

dim3 gridSize(1);
dim3 blockSize(ARRAY_SIZE, ARRAY_SIZE);

__host__ void cudaAdd(float *a, float *b, float *c) {
    hipMemcpy(devA, a, ARRAY_MEM_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(devB, b, ARRAY_MEM_SIZE, hipMemcpyHostToDevice);
    kernMatAdd<<<gridSize, blockSize>>>(devA, devB, devC);
    hipMemcpy(c, devC, ARRAY_MEM_SIZE, hipMemcpyDeviceToHost);
}

__host__ void cudaSub(float *a, float *b, float *c) {
    hipMemcpy(devA, a, ARRAY_MEM_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(devB, b, ARRAY_MEM_SIZE, hipMemcpyHostToDevice);
    kernMatSub<<<gridSize, blockSize>>>(devA, devB, devC);
    hipMemcpy(c, devC, ARRAY_MEM_SIZE, hipMemcpyDeviceToHost);
}

__host__ void cudaMul(float *a, float *b, float *c) {
    hipMemcpy(devA, a, ARRAY_MEM_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(devB, b, ARRAY_MEM_SIZE, hipMemcpyHostToDevice);
    kernMatMul<<<gridSize, blockSize>>>(devA, devB, devC);
    hipMemcpy(c, devC, ARRAY_MEM_SIZE, hipMemcpyDeviceToHost);
}
