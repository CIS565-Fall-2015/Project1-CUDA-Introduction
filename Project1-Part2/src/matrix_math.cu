#include "hip/hip_runtime.h"
#include "matrix_math.h"

float *dev_mat_A;
float *dev_mat_B;
float *dev_mat_C;

hipEvent_t start, stop;

void CUDA_matrix_math::initialize() {
	hipMalloc((void**)&dev_mat_A, sizeof(float) * 25);
	hipMalloc((void**)&dev_mat_B, sizeof(float) * 25);
	hipMalloc((void**)&dev_mat_C, sizeof(float) * 25);
}

void CUDA_matrix_math::teardown() {
	hipFree(dev_mat_A);
	hipFree(dev_mat_B);
	hipFree(dev_mat_C);
}

__global__ void mat_add(float *A, float *B, float *C) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j * 5;
	C[index] = A[index] + B[index];
}

__global__ void mat_sub(float *A, float *B, float *C) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j * 5;
	C[index] = A[index] - B[index];
}

__global__ void mat_mul(float *A, float *B, float *C) {
	int corner_x = blockIdx.x * blockDim.x;
	int corner_y = blockIdx.y * blockDim.y;
	int index = (corner_x + threadIdx.x) + (corner_y + threadIdx.y) * 5;
	float dot_product = 0.0f;

	// all values are + blockIdx.x * bloxkDim.x + blockIdx.x + blockIdx.y
	// 0  1  2  3  4
	// 5  6  7  8  9
	// 10 11 12 13 14
	// 15 16 17 18 19
	// 20 21 22 23 24
	int local_index = threadIdx.x + threadIdx.y * 5;
	int corner_index = corner_x + corner_y * 5;
	int col_index = local_index % 5 + corner_index;
	int row_index = (local_index / 5) * 5 + corner_index;
	for (int i = 0; i < 5; i++) {
		dot_product += A[row_index] * B[col_index];
		col_index += 5;
		row_index += 1;
	}

	C[index] = dot_product;
}

static void setup_timer_events() {
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

static float teardown_timer_events() {
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return milliseconds;
}

void CUDA_matrix_math::cuda_mat_add(float *A, float *B, float *C) {
	hipMemcpy(dev_mat_A, A, 25 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_B, B, 25 * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(5, 5);
	dim3 dimGrid(1, 1);

	setup_timer_events();
	hipEventRecord(start);
	mat_add <<<dimGrid, dimBlock >>>(dev_mat_A, dev_mat_B, dev_mat_C);
	hipEventRecord(stop);

	float time = teardown_timer_events();
	printf("Addition operation took about %f milliseconds.\n", time);

	hipMemcpy(C, dev_mat_C, 25 * sizeof(float), hipMemcpyDeviceToHost);
}

void CUDA_matrix_math::cuda_mat_sub(float *A, float *B, float *C) {
	hipMemcpy(dev_mat_A, A, 25 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_B, B, 25 * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(5, 5);
	dim3 dimGrid(1, 1);

	setup_timer_events();
	hipEventRecord(start);
	mat_sub <<<dimGrid, dimBlock >>>(dev_mat_A, dev_mat_B, dev_mat_C);
	hipEventRecord(stop);

	float time = teardown_timer_events();
	printf("Subtraction operation took about %f milliseconds.\n", time);

	hipMemcpy(C, dev_mat_C, 25 * sizeof(float), hipMemcpyDeviceToHost);
}

void CUDA_matrix_math::cuda_mat_mul(float *A, float *B, float *C) {
	hipMemcpy(dev_mat_A, A, 25 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_B, B, 25 * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(5, 5);
	dim3 dimGrid(1, 1);

	setup_timer_events();
	hipEventRecord(start);
	mat_mul <<<dimGrid, dimBlock >>>(dev_mat_A, dev_mat_B, dev_mat_C);
	hipEventRecord(stop);

	float time = teardown_timer_events();
	printf("Multiplication operation took about %f milliseconds.\n", time);
	hipMemcpy(C, dev_mat_C, 25 * sizeof(float), hipMemcpyDeviceToHost);
}