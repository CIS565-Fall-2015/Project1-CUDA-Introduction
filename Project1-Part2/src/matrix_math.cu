#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "matrix_math.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


/***********************************************
* Host state *
***********************************************/
int hst_width;

float *hst_mat_a;
float *hst_mat_b;
float *hst_mat_c;

/***********************************************
* Device state *
***********************************************/


dim3 block_dim;
dim3 grid_dim;

float *dev_mat_a;
float *dev_mat_b;
float *dev_mat_c;


/******************
* initialization *
******************/

void MatrixMath::initialization(int mat_width) {
	hst_width = mat_width;
	// TODO: Not sure on how to set this up, cause we are using a 1D array, not 2D. Think this is valid
	block_dim = dim3(mat_width * mat_width);
	grid_dim = dim3(1, 1);

	// Is it this memory I am going to have to move?
	// use malloc then move this to the device
	//what am i initiallizing hte values too here?
	hst_mat_a = (float*)malloc((mat_width * mat_width) * sizeof(float));
	// TODO: Error check

	hst_mat_b = (float*)malloc((mat_width * mat_width) * sizeof(float));
	// TODO: Error check

	hst_mat_c = (float*)malloc((mat_width * mat_width) * sizeof(float));
	// TODO: Error check

	//think I only want to allocate on the device when doing cuda

	hipMalloc((void**)&dev_mat_a, (mat_width * mat_width) * sizeof(float));
	checkCUDAErrorWithLine("hipMalloc dev_mat_a failed!");

	hipMalloc((void**)&dev_mat_b, (mat_width * mat_width) * sizeof(float));
	checkCUDAErrorWithLine("hipMalloc dev_mat_b failed!");

	hipMalloc((void**)&dev_mat_c, (mat_width * mat_width) * sizeof(float));
	checkCUDAErrorWithLine("hipMalloc dev_mat_c failed!");
}

void MatrixMath::cleanup() {
	// Why is freeing the host memory throwing errors?
	//free(hst_mat_a);
	//free(hst_mat_b);
	//free(hst_mat_c);

	hipFree(dev_mat_a);
	hipFree(dev_mat_b);
	hipFree(dev_mat_c);
}

__global__ void kern_mat_add(float *A, float *B, float *C, int width) {
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

__global__ void kern_mat_sub(float *A, float *B, float *C, int width) {
	int i = threadIdx.x;
	C[i] = A[i] - B[i];
}

__global__ void kern_mat_mul(float *A, float *B, float *C, int width) {
	//int i = threadIdx.x;
	int i = threadIdx.x % width;
	int j = threadIdx.x / width;
	float Ci = 0.0f;
	//this is totally wrong?
	for (int k = 0; k < width; k++) {
		float Ai = A[j * width + k];
		float Bi = B[k * width + i];

		Ci += Ai * Bi;
	}
	C[j * width + i] = Ci;
}

void MatrixMath::mat_add(float *A, float *B, float *C) {
	// first copy to device memory, then envoke kernel function
	hipMemcpy(dev_mat_a, A, (hst_width * hst_width) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_b, B, (hst_width * hst_width) * sizeof(float), hipMemcpyHostToDevice);
	kern_mat_add<<<grid_dim, block_dim>>>(dev_mat_a, dev_mat_b, dev_mat_c, hst_width);
	hipMemcpy(C, dev_mat_c, (hst_width * hst_width) * sizeof(float), hipMemcpyDeviceToHost);
}

void MatrixMath::mat_sub(float *A, float *B, float *C) {
	hipMemcpy(dev_mat_a, A, (hst_width * hst_width) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_b, B, (hst_width * hst_width) * sizeof(float), hipMemcpyHostToDevice);
	kern_mat_sub<<<grid_dim, block_dim>>>(dev_mat_a, dev_mat_b, dev_mat_c, hst_width);
	hipMemcpy(C, dev_mat_c, (hst_width * hst_width) * sizeof(float), hipMemcpyDeviceToHost);
}

void MatrixMath::mat_mul(float *A, float *B, float *C) {
	hipMemcpy(dev_mat_a, A, (hst_width * hst_width) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat_b, B, (hst_width * hst_width) * sizeof(float), hipMemcpyHostToDevice);
	kern_mat_mul<<<grid_dim, block_dim>>>(dev_mat_a, dev_mat_b, dev_mat_c, hst_width);
	hipMemcpy(C, dev_mat_c, (hst_width * hst_width) * sizeof(float), hipMemcpyDeviceToHost);
}

void MatrixMath::print_mat(float *mat, int width) {
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < width; j++) {
			fprintf(stdout, "%f, ", mat[(i * width) + j]);
		}
		fprintf(stdout, "\n");
	}
}

/*
	This is where we will run tests to confirm the functions work
*/
void MatrixMath::run_tests() {
	float A[] = {
		9.0f, 10.0f, 2.0f, 1.0f, 7.5f,
		2.0f, 1.0f, 1.0f, 1.0f, 1.0f,
		1.1f, 3.0f, 1.0f, 20.0f, 13.0f,
		6.6f, 3.0f, 0.0f, 1.0f, 9.0f,
		2.0f, 4.0f, 8.0f, 4.5f, 0.0f,
	};
	hst_mat_a = A;

	float B[] = {
		2.0f, 1.0f, 3.0f, 5.0f, 9.0f,
		0.5f, 0.1f, 19.0f, 2.0f, 12.0f,
		5.0f, 2.3f, 8.0f, 2.0f, 13.0f,
		6.0f, 4.5f, 9.0f, 1.0f, 0.75f,
		11.0f, 11.0f, 7.8f, 22.0f, 1.0f,
	};
	hst_mat_b = B;

	float C[] = {
		0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
		0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
		0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
		0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
		0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
	};
	hst_mat_c = C;

	fprintf(stdout, "Running matrix addition, subtraction, and multiplcation tests...\n\n");

	fprintf(stdout, "Matrix A:\n");
	MatrixMath::print_mat(A, hst_width);

	fprintf(stdout, "\n\n");

	fprintf(stdout, "Matrix B:\n");
	MatrixMath::print_mat(B, hst_width);

	fprintf(stdout, "\n\n");

	fprintf(stdout, "Addition Test A + B = \n");
	MatrixMath::mat_add(A, B, C);
	MatrixMath::print_mat(C, hst_width);

	fprintf(stdout, "\n\n");

	fprintf(stdout, "Subtraction Test A - B = \n");
	MatrixMath::mat_sub(A, B, C);
	MatrixMath::print_mat(C, hst_width);

	fprintf(stdout, "\n\n");

	fprintf(stdout, "Multiplication Test A * B = \n");
	MatrixMath::mat_mul(A, B, C);
	MatrixMath::print_mat(C, hst_width);

	cleanup();
}