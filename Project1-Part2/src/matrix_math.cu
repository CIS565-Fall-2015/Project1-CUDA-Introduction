#include "hip/hip_runtime.h"
#include "matrix_math.h"


#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
 * Check for CUDA errors; print and exit if there was a problem.
 */
void checkCUDAError(const char *msg, int line = -1) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        if (line >= 0) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int N = 25;
float* dev_mat_a;
float* dev_mat_b;
float* dev_mat_c;

/**
 * Initialize memory, update some globals
 */
void Matrix_Math::initialize() {
    int N = 25;
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

    hipMalloc((void**)&hst_mat, N * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc hst_mat failed!");

    hipMalloc((void**)&dev_mat_a, N * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc dev_mat_a failed!");

    hipMalloc((void**)&dev_mat_b, N * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc dev_mat_b failed!");

    hipMalloc((void**)&dev_mat_c, N * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc dev_mat_c failed!");

    hipDeviceSynchronize();
}

void Matrix_Math::cleanUp() {
    hipFree(dev_mat_a);
    hipFree(dev_mat_b);
    hipFree(dev_mat_c);
}

__global__ void mat_add(float *A, float *B, float *C) {
}

__global__ void mat_sub(float *A, float *B, float *C) {
}

__global__ void mat_mul(float *A, float *B, float *C) {
}