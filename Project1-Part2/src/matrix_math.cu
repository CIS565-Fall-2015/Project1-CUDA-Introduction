#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>

#include "matrix_math.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)
#define mat_size 5
#define block_size 1
/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/


//dim3 threadsPerBlock(blockSize);

float *dev_MA;
float *dev_MB;
float *dev_MC;

void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
/************************************************************************
*initialize two 5x5 matrices on the host and two on the device *********
************************************************************************/
void NMatrix::initialization(float *hst_MA, float *hst_MB) {

	float size_z = mat_size * mat_size * sizeof(float);

	hipMalloc((void**)&dev_MA, size_z);
	checkCUDAErrorWithLine("hipMalloc dev_MA failed!");

	hipMalloc((void**)&dev_MB, size_z);
	checkCUDAErrorWithLine("hipMalloc dev_MB failed!");
	hipMalloc((void**)&dev_MC, size_z);
	checkCUDAErrorWithLine("hipMalloc dev_MC failed!");

	hipMemcpy(dev_MA, hst_MA, size_z, hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy hst_MA failed!");

	hipMemcpy(dev_MB, hst_MB, size_z, hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy hst_MA failed!");
}
/*implement function*/
__global__ void Add(float* A, float *B, float *C){
//__global__ void Add(float A[mat_size][mat_size], float B[mat_size][mat_size], float C[mat_size][mat_size]){
	int i = threadIdx.x;
	int j = threadIdx.y;
	int index = i*mat_size + j;
	if (i < mat_size && j < mat_size)
		//C[i][j] = A[i][j] + B[i][j];
		C[index] = A[index] + B[index];
	
}

__global__ void Sub(float *A, float* B, float *C){
	int id = threadIdx.x*mat_size+threadIdx.y;
	C[id] = A[id] - B[id];
}

__global__ void Mul(float *A, float *B, float *C){
	int id = threadIdx.x*mat_size + threadIdx.y;

	for (int k = 0; k < mat_size; k++){
		//C[i][j]+ = A[i][k] * B[k][j];
	    //i=threadIdx.x,j=threadIdx.y
		C[id] += A[threadIdx.x*mat_size + k] * B[k*mat_size + threadIdx.y];
    }
	
}
void NMatrix::mat_add(float *hst_A, float * hst_B, float* hst_C){
	

	dim3 threadsPerBlock(mat_size, mat_size);
	Add <<< block_size, threadsPerBlock >> >(dev_MA, dev_MB, dev_MC);
	hipMemcpy(hst_C, dev_MC,  mat_size*mat_size*sizeof(float), hipMemcpyDeviceToHost);//destination,source,
	endMAtrix();
}
void NMatrix::mat_sub(float  *hst_A, float *hst_B, float* hst_C){
	
	dim3 threadsPerBlock(mat_size, mat_size);
	Sub <<< block_size, threadsPerBlock >>>(dev_MA, dev_MB, dev_MC);
	hipMemcpy(hst_C, dev_MC, mat_size*mat_size*sizeof(float), hipMemcpyDeviceToHost);//destination,source,
	endMAtrix();
}

void NMatrix::mat_mul(float  *hst_A, float * hst_B, float* hst_C){
//	C = A * B;
	
	dim3 threadsPerBlock(mat_size, mat_size);
	Mul <<< block_size, threadsPerBlock >>>(dev_MA, dev_MB, dev_MC);
	hipMemcpy(hst_C, dev_MC, mat_size*mat_size*sizeof(float), hipMemcpyDeviceToHost);//destination,source,
	endMAtrix();
}

void NMatrix::endMAtrix()
{
	hipFree(dev_MA);
	hipFree(dev_MB);
	hipFree(dev_MC);
}