#include "hip/hip_runtime.h"
#include "matrix_math.h"
#include<hip/hip_runtime.h>

float *dev_A,*dev_B,*dev_C;

void Matrix_Math::initiate(int size,float *A,float *B){
	hipMalloc((void**)&dev_A, size*size * sizeof(float));
	hipMalloc((void**)&dev_B, size*size * sizeof(float));
	hipMalloc((void**)&dev_C, size*size * sizeof(float));
	hipMemcpy(dev_A,A,size*size*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_B,B,size*size*sizeof(float),hipMemcpyHostToDevice);
}

void Matrix_Math::copyBack(int size,float *C){
	hipMemcpy(C,dev_C,size*size*sizeof(float),hipMemcpyDeviceToHost);
}

__global__ void mat_add(int N,float *A,float *B,float *C){
	int index=blockIdx.x*blockDim.x*blockDim.y+blockDim.x*threadIdx.y+threadIdx.x;
	if(index<N*N)
		C[index]=A[index]+B[index];
}

__global__ void mat_sub(int N,float *A,float *B,float *C){
	int index=blockIdx.x*blockDim.x*blockDim.y+blockDim.x*threadIdx.y+threadIdx.x;
	if(index<N*N)
		C[index]=A[index]-B[index];
}

__global__ void mat_mul(int n,float *A,float *B,float *C){
	int index=blockIdx.x*blockDim.x*blockDim.y+blockDim.x*threadIdx.y+threadIdx.x;
	int result=0;
	for(int i=0;i<n;++i){
		result+=A[blockDim.x*threadIdx.y+i]*B[blockDim.x*i+threadIdx.x];
	}
	C[index]=result;
}

float Matrix_Math::add(int inputSize,int blockSize,float *A,float *B,float *C){
	hipEvent_t start, stop;
	initiate(inputSize,A,B);
	dim3 blockNum=(1,1+inputSize*inputSize/blockSize/blockSize);
	dim3 block(blockSize,blockSize);
	hipEventCreate(&start);
	mat_add<<<blockNum,block>>>(inputSize,dev_A,dev_B,dev_C);
	hipEventCreate(&stop);
	copyBack(inputSize,C);
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	return hipEventElapsedTime(&milliseconds, start, stop);
}

float Matrix_Math::sub(int inputSize,int blockSize,float *A,float *B,float *C){
	hipEvent_t start, stop;
	initiate(inputSize,A,B);
	dim3 blockNum=(1,1+inputSize*inputSize/blockSize/blockSize);
	dim3 block(blockSize,blockSize);
	hipEventCreate(&start);
	mat_sub<<<blockNum,block>>>(inputSize,dev_A,dev_B,dev_C);
	hipEventCreate(&stop);
	copyBack(inputSize,C);
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	return hipEventElapsedTime(&milliseconds, start, stop);
}

float Matrix_Math::mul(int inputSize,int blockSize,float *A,float *B,float *C){
	hipEvent_t start, stop;
	initiate(inputSize,A,B);
	dim3 blockNum=(1,1+inputSize*inputSize/blockSize/blockSize);
	dim3 block(blockSize,blockSize);
	hipEventCreate(&start);
	mat_mul<<<blockNum,block>>>(inputSize,dev_A,dev_B,dev_C);
	hipEventCreate(&stop);
	copyBack(inputSize,C);
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	return hipEventElapsedTime(&milliseconds, start, stop);
}