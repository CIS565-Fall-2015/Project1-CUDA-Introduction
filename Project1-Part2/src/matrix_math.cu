#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "matrix_math.h"
#include "utilityCore.hpp"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


/*****************
* Configuration *
*****************/

#define N 5

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

float *dev_mat1;
float *dev_mat2;
float *hos_mat1;
float *hos_mat2;

/******************
* Setup & tear down *
******************/

/**
* Initialize memory, update some globals
*/
void MMath::init() {

	hipMalloc((void**)&dev_mat1, N * N * sizeof(float));
	checkCUDAErrorWithLine("hipMalloc dev_mat1 failed!");

	hipMalloc((void**)&dev_mat2, N * N * sizeof(float));
	checkCUDAErrorWithLine("hipMalloc dev_mat2 failed!");

	hos_mat1 = (float *)malloc(N * N * sizeof(float));
	hos_mat2 = (float *)malloc(N * N * sizeof(float));

	hipMemcpy(hos_mat1, dev_mat1, N * N * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy hos_mat1 failed!");

	hipMemcpy(hos_mat2, dev_mat2, N * N * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy hos_mat2 failed!");

	//test();
	test2();
	test3();

	/*
	1 Block, 5x5 Thread: A:0.006976, S:0.004736, M:0.007200
	5 Block, 5x1 Thread: A:0.004704, S:0.004672, M:0.007264
	*/
}

void MMath::terminate() {
	hipFree(dev_mat1);
	hipFree(dev_mat2);
	free(hos_mat1);
	free(hos_mat2);
}


/******************
* Kernels *
******************/

/**
* Addition
*/

__global__ void mat_add(float *A, float *B, float *C){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	C[j*N + i] = A[j*N + i] + B[j*N + i];
}

/**
* Subtraction
*/
__global__ void mat_sub(float *A, float *B, float *C){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	C[j*N + i] = A[j*N + i] - B[j*N + i];
}

/**
* Multiplication
*/
__global__ void mat_mul(float *A, float *B, float *C){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	float p = 0;

	for (int k = 0; k < N; k++){
		p += A[j*N + k] * B[k*N + i];
	}

	C[j*N + i] = p;
}

void MMath::test(){
	for (int i = 0; i < N*N; i++){
		hos_mat1[i] = 1.0f;
		hos_mat2[i] = 2.0f;
	}

	float *dev_matp;
	float *hos_matp;

	hipMalloc((void**)&dev_matp, N * N * sizeof(float));
	checkCUDAErrorWithLine("hipMalloc dev_matp failed!");

	hos_matp = (float *)malloc(N * N * sizeof(float));

	// Test add
	hipMemcpy(dev_mat1, hos_mat1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2, hos_mat2, N * N * sizeof(float), hipMemcpyHostToDevice);

	dim3 gridSize(1, 1);
	dim3 blockSize(N, N);

	printf("Addition result:\n");
	mat_add<<<gridSize, blockSize>>>(dev_mat1, dev_mat2, dev_matp);

	hipMemcpy(hos_matp, dev_matp, N * N * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy hos_matp failed!");

	for (int i = 0; i < N*N; i++) {
		printf("%f ", hos_matp[i]);
		printf("\n");
	}

	// Test sub
	hipMemcpy(dev_mat1, hos_mat1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2, hos_mat2, N * N * sizeof(float), hipMemcpyHostToDevice);

	printf("Subtraction result:\n");
	mat_sub<<<gridSize, blockSize>>>(dev_mat1, dev_mat2, dev_matp);

	hipMemcpy(hos_matp, dev_matp, N * N * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy hos_matp failed!");

	for (int i = 0; i < N*N; i++) {
		printf("%f ", hos_matp[i]);
		printf("\n");
	}

	// Test mul
	hipMemcpy(dev_mat1, hos_mat1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2, hos_mat2, N * N * sizeof(float), hipMemcpyHostToDevice);

	printf("Multiplication result:\n");
	mat_mul<<<gridSize, blockSize>>>(dev_mat1, dev_mat2, dev_matp);

	hipMemcpy(hos_matp, dev_matp, N * N * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy hos_matp failed!");

	for (int i = 0; i < N*N; i++) {
		printf("%f ", hos_matp[i]);
		printf("\n");
	}

	hipFree(dev_matp);
	free(hos_matp);
}

void MMath::test2(){
	for (int i = 0; i < N*N; i++){
		hos_mat1[i] = 1.0f;
		hos_mat2[i] = 2.0f;
	}

	float *dev_matp;
	float *hos_matp;

	hipMalloc((void**)&dev_matp, N * N * sizeof(float));
	hos_matp = (float *)malloc(N * N * sizeof(float));

	hipEvent_t start, stop;

	// --------------------------------------------------------------------------------

	dim3 gridSize(1, 1);
	dim3 blockSize(N, N);

	// Test add
	hipMemcpy(dev_mat1, hos_mat1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2, hos_mat2, N * N * sizeof(float), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	mat_add << <gridSize, blockSize >> >(dev_mat1, dev_mat2, dev_matp);
	hipEventRecord(stop);

	hipMemcpy(hos_matp, dev_matp, N * N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float msAdd1 = 0;
	hipEventElapsedTime(&msAdd1, start, stop);

	// Test sub
	hipMemcpy(dev_mat1, hos_mat1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2, hos_mat2, N * N * sizeof(float), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	mat_sub << <gridSize, blockSize >> >(dev_mat1, dev_mat2, dev_matp);
	hipEventRecord(stop);

	hipMemcpy(hos_matp, dev_matp, N * N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float msSub1 = 0;
	hipEventElapsedTime(&msSub1, start, stop);

	// Test mul
	hipMemcpy(dev_mat1, hos_mat1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2, hos_mat2, N * N * sizeof(float), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	mat_mul << <gridSize, blockSize >> >(dev_mat1, dev_mat2, dev_matp);
	hipEventRecord(stop);

	hipMemcpy(hos_matp, dev_matp, N * N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float msMul1 = 0;
	hipEventElapsedTime(&msMul1, start, stop);

	hipFree(dev_matp);
	free(hos_matp);

	printf("1 Block, 5x5 Thread: A:%f, S:%f, M:%f\n", msAdd1, msSub1, msMul1);
}

void MMath::test3(){
	for (int i = 0; i < N*N; i++){
		hos_mat1[i] = 1.0f;
		hos_mat2[i] = 2.0f;
	}

	float *dev_matp;
	float *hos_matp;

	hipMalloc((void**)&dev_matp, N * N * sizeof(float));
	hos_matp = (float *)malloc(N * N * sizeof(float));

	hipEvent_t start, stop;

	// --------------------------------------------------------------------------------

	dim3 gridSize(1, 5);
	dim3 blockSize(N, 1);

	// Test add
	hipMemcpy(dev_mat1, hos_mat1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2, hos_mat2, N * N * sizeof(float), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	mat_add << <gridSize, blockSize >> >(dev_mat1, dev_mat2, dev_matp);
	hipEventRecord(stop);

	hipMemcpy(hos_matp, dev_matp, N * N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float msAdd1 = 0;
	hipEventElapsedTime(&msAdd1, start, stop);

	// Test sub
	hipMemcpy(dev_mat1, hos_mat1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2, hos_mat2, N * N * sizeof(float), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	mat_sub << <gridSize, blockSize >> >(dev_mat1, dev_mat2, dev_matp);
	hipEventRecord(stop);

	hipMemcpy(hos_matp, dev_matp, N * N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float msSub1 = 0;
	hipEventElapsedTime(&msSub1, start, stop);

	// Test mul
	hipMemcpy(dev_mat1, hos_mat1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2, hos_mat2, N * N * sizeof(float), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	mat_mul << <gridSize, blockSize >> >(dev_mat1, dev_mat2, dev_matp);
	hipEventRecord(stop);

	hipMemcpy(hos_matp, dev_matp, N * N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float msMul1 = 0;
	hipEventElapsedTime(&msMul1, start, stop);

	hipFree(dev_matp);
	free(hos_matp);

	printf("5 Block, 5x1 Thread: A:%f, S:%f, M:%f\n", msAdd1, msSub1, msMul1);
}