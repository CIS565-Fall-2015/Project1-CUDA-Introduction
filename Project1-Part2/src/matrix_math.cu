#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
//#include "utilityCore.hpp"
#include "matrix_math.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
 * Check for CUDA errors; print and exit if there was a problem.
 */
void checkCUDAError(const char *msg, int line = -1) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        if (line >= 0) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


/*****************
 * Configuration *
 *****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 1024


/***********************************************
 * Kernel state  *
 ***********************************************/

dim3 threadsPerBlock(blockSize);
dim3 fullBlocksPerGrid((25 + blockSize - 1) / blockSize);

float *dev_A;
float *dev_B;
float *dev_C;


/******************
 * init *
 ******************/


/**
 * Initialize memory, update some globals
 */
void Matrix_Math::init() {
    //dim3 fullBlocksPerGrid(1);

    hipMalloc((void**)&dev_A, 25 * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc dev_A failed!");

    hipMalloc((void**)&dev_B, 25 * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc dev_B failed!");

    hipMalloc((void**)&dev_C, 25 * sizeof(float));
    checkCUDAErrorWithLine("hipMalloc dev_C failed!");

//    hipMemcpy(dev_A, hst_A, 25 * sizeof(float), hipMemcpyHostToDevice);
//    checkCUDAErrorWithLine("hipMemcpy hst_A to dev_A failed!");
//
//    hipMemcpy(dev_B, hst_B, 25 * sizeof(float), hipMemcpyHostToDevice);
//    checkCUDAErrorWithLine("hipMemcpy hst_B to dev_B failed!");

    //hipDeviceSynchronize();
}

/******************
 * Matrix_Math *
 ******************/

__global__ void mat_add(float *dev_A, float *dev_B, float *dev_C) {
    // TODO: implement updateAccArray.
    // This function body runs once on each CUDA thread.
    // To avoid race conditions, each instance should only write ONE value to `acc`!
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	dev_C[index] = dev_A[index] + dev_B[index];
}

__global__ void mat_sub(float *dev_A, float *dev_B, float *dev_C) {
    // TODO: implement updateAccArray.
    // This function body runs once on each CUDA thread.
    // To avoid race conditions, each instance should only write ONE value to `acc`!
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	dev_C[index] = dev_A[index] - dev_B[index];
}

__global__ void mat_mul(float *dev_A, float *dev_B, float *dev_C) {
    // TODO: implement updateAccArray.
    // This function body runs once on each CUDA thread.
    // To avoid race conditions, each instance should only write ONE value to `acc`!
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	int row = index/5;
	int col = index%5;
	float result = 0;
	for (int i=0; i<5; i++){
		result = result + dev_A[(row*5) + i] * dev_B[(i*5) + col];
	}
	dev_C[index] = result;
}

/******************
 * Matrix_Math *
 ******************/
float Matrix_Math::add(float* A, float* B, float* C){
    init();
	hipMemcpy(dev_A, A, 25 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, 25 * sizeof(float), hipMemcpyHostToDevice);

	float time = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	mat_add<<<fullBlocksPerGrid, blockSize >>>(dev_A, dev_B, dev_C);
	hipEventRecord(stop);

	hipMemcpy(C, dev_C, 25 * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy dev_C to hst_C failed!");

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	return time;
}

float Matrix_Math::sub(float* A, float* B, float* C){
    init();
    hipMemcpy(dev_A, A, 25 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, 25 * sizeof(float), hipMemcpyHostToDevice);

    float time = 0;
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	mat_sub<<< fullBlocksPerGrid, blockSize >>>(dev_A, dev_B, dev_C);
	hipEventRecord(stop);

	hipMemcpy( C , dev_C, 25 * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy dev_C to hst_C failed!");

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	return time;
}

float Matrix_Math::mul(float* A, float* B, float* C){
    init();
	hipMemcpy(dev_A, A, 25 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, 25 * sizeof(float), hipMemcpyHostToDevice);

    float time = 0;
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	mat_mul<<< fullBlocksPerGrid, blockSize >>>(dev_A, dev_B, dev_C);
	hipEventRecord(stop);

	hipMemcpy( C, dev_C, 25 * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy dev_C to hst_C failed!");

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	return time;
}
